#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <inttypes.h>


#include "utils.hpp"
#include "update.hpp"
#include "cuStinger.hpp"

#include "operators.cuh"
#include "algs.cuh"
#include "streaming_bc/bc.cuh"
#include "streaming_bc/bc_tree.cuh"


using namespace std;

namespace cuStingerAlgs {


void StreamingBC::Init(cuStinger& custing)
{
	forest = createBcForest(custing.nv, nr);

	for (length_t k = 0; k < nr; k++)
	{
		forest->trees_h[k]->queue.Init(custing.nv);
	}

	host_deltas = new float[custing.nv];
	cusLB = new cusLoadBalance(custing.nv);

	// Keep a device copy of the array forest->trees_d
	trees_d = (bcTree**) allocDeviceArray(nr, sizeof(bcTree*));
	copyArrayHostToDevice(forest->trees_d, trees_d, nr, sizeof(bcTree*));

	if (nr == custing.nv)
	{
		approx = false;
	}
	Reset();
}


void StreamingBC::Reset()
{
	for (length_t k = 0; k < nr; k++)
	{
		bcTree *hostBcTree = forest->trees_h[k];
		hostBcTree->queue.resetQueue();
		hostBcTree->currLevel = 0;

		if (approx) {
			hostBcTree->root = rand() % forest->nv;
		} else {
			hostBcTree->root = k;
		}

		// initialize all offsets to zero
		for (int i = 0; i < hostBcTree->nv; i++)
		{
			hostBcTree->offsets[i] = 0;
		}

		SyncDeviceWithHost(k);
	}
}

// Must pass in a root node vertex id, and a pointer to bc values (of length custing.nv)
void StreamingBC::setInputParameters(float *bc_array)
{
	bc = bc_array;
}

void StreamingBC::Release()
{
	delete cusLB;
	delete[] host_deltas;
	destroyBcForest(forest, nr);
	freeDeviceArray(trees_d);
}


void StreamingBC::Run(cuStinger& custing)
{
	for (length_t k = 0; k < nr; k++)
	{
		RunBfsTraversal(custing, k);
		DependencyAccumulation(custing, k);
	}
}


void StreamingBC::RunBfsTraversal(cuStinger& custing, length_t k)
{
	bcTree *deviceBcTree = forest->trees_d[k];
	bcTree *hostBcTree = forest->trees_h[k];

	// Clear out array values first
	allVinG_TraverseVertices<bcOperator::setupArrays>(custing, deviceBcTree);
	hostBcTree->queue.enqueueFromHost(hostBcTree->root);
	SyncDeviceWithHost(k);

	// set d[root] <- 0
	int zero = 0;
	copyArrayHostToDevice(&zero, hostBcTree->d + hostBcTree->root,
		1, sizeof(length_t));

	// set sigma[root] <- 1
	int one = 1;
	copyArrayHostToDevice(&one, hostBcTree->sigma + hostBcTree->root,
		1, sizeof(length_t));

	length_t prevEnd = 1;
	hostBcTree->offsets[0] = 1;

	while( hostBcTree->queue.getActiveQueueSize() > 0)
	{
		allVinA_TraverseEdges_LB<bcOperator::bcExpandFrontier>(custing,
			deviceBcTree, *cusLB, hostBcTree->queue);
		SyncHostWithDevice(k);

		// Update cumulative offsets from start of queue
		hostBcTree->queue.setQueueCurr(prevEnd);

		vertexId_t level = getLevel(k);
		hostBcTree->offsets[level + 1] = hostBcTree->queue.getActiveQueueSize() + hostBcTree->offsets[level];

		prevEnd = hostBcTree->queue.getQueueEnd();

		hostBcTree->currLevel++;
		SyncDeviceWithHost(k);
	}
}


void StreamingBC::DependencyAccumulation(cuStinger& custing, length_t k)
{
	bcTree *deviceBcTree = forest->trees_d[k];
	bcTree *hostBcTree = forest->trees_h[k];

	// Iterate backwards through depths, starting from 2nd deepest frontier
	// Begin with the 2nd deepest frontier as the active queue
	hostBcTree->currLevel -= 2;
	SyncDeviceWithHost(k);

	while (getLevel(k) >= 0)
	{
		length_t start = hostBcTree->offsets[getLevel(k)];
		length_t end = hostBcTree->offsets[getLevel(k) + 1];

		// // set queue start and end so the queue holds all nodes in one frontier
		hostBcTree->queue.setQueueCurr(start);
		hostBcTree->queue.setQueueEnd(end);
		hostBcTree->queue.SyncDeviceWithHost();
		SyncDeviceWithHost(k);

		// Now, run the macro for all outbound edges over this queue
		allVinA_TraverseEdges_LB<bcOperator::dependencyAccumulation>(custing,
			deviceBcTree, *cusLB, hostBcTree->queue);
		SyncHostWithDevice(k);

		hostBcTree->currLevel -= 1;
		SyncDeviceWithHost(k);
	}

	// Now, copy over delta values to host
	copyArrayDeviceToHost(hostBcTree->delta, host_deltas, hostBcTree->nv, sizeof(float));

	// // Finally, update the bc values
	for (vertexId_t w = 0; w < hostBcTree->nv; w++)
	{
		if (w != hostBcTree->root)
		{
			bc[w] += host_deltas[w];
		}
	}
}


void StreamingBC::InsertEdge(cuStinger& custing, vertexId_t src, vertexId_t dst)
{
	vertexId_t *diffs_h = new vertexId_t[nr];
	getDepthDifferences(custing, src, dst, nr, diffs_h, trees_d);

	SyncHostWithDevice();  // copy the updated ulow, uhigh assignments to host

	// cases where ulow = dst and uhigh = src
	vertexId_t adj = 0;
	vertexId_t nonadj = 0;
	// cases where ulow = src and uhigh = dst
	vertexId_t adjRev = 0;
	vertexId_t nonadjRev = 0;

	vertexId_t size = 0;

	vertexId_t* caseArray_h = buildCaseArray(diffs_h, nr, size, adj, nonadj,
		adjRev, nonadjRev);

	vertexId_t* caseArray_d = (vertexId_t*) allocDeviceArray(size, sizeof(vertexId_t));
	copyArrayHostToDevice(caseArray_h, caseArray_d, size, sizeof(vertexId_t));
	// Now, run an operator to handle each case


	delete[] diffs_h;
	delete[] caseArray_h;
}


void StreamingBC::RemoveEdge(cuStinger& custing, vertexId_t src, vertexId_t dst)
{

}


void getDepthDifferences(cuStinger& custing, vertexId_t src, vertexId_t dst,
	length_t numRoots, vertexId_t* diffs_h, bcTree** trees_d)
{
	// TODO: optimize these device allocations
	depthDiffs *dDiffs_h = new depthDiffs;
	depthDiffs *dDiffs_d = (depthDiffs*) allocDeviceArray(1, sizeof(depthDiffs));

	dDiffs_h->trees_d = trees_d;
	dDiffs_h->numRoots = numRoots;
	dDiffs_h->src = src;
	dDiffs_h->dst = dst;

	// allocate space on device for diffs_d
	dDiffs_h->diffs_d = (vertexId_t*) allocDeviceArray(numRoots, sizeof(vertexId_t));
	// copy contents of host struct onto device struct
	copyArrayHostToDevice(dDiffs_h, dDiffs_d, 1, sizeof(depthDiffs));

	// we need an array of "roots" that we can use with our operator
	vertexId_t* rootArray_h = new vertexId_t[numRoots];
	for (int k = 0; k < numRoots; k++) {
		rootArray_h[k] = k;
	}

	// need the same array on the device
	vertexId_t* rootArray_d = (vertexId_t*) allocDeviceArray(numRoots,
		sizeof(vertexId_t));

	copyArrayHostToDevice(rootArray_h, rootArray_d, numRoots,
		sizeof(vertexId_t));

	// now, use a streaming operator to get the depth differences and ulow and uhigh assignments
	allVinA_TraverseVertices<bcOperator::preprocessEdge>(custing,
		(void*) dDiffs_d, rootArray_d, numRoots);

	// store the results in diffs_h
	copyArrayDeviceToHost(dDiffs_h->diffs_d, diffs_h, numRoots,
		sizeof(vertexId_t));

	// Free device memory
	freeDeviceArray(rootArray_d);
	freeDeviceArray(dDiffs_d);
	freeDeviceArray(dDiffs_h->diffs_d);

	// Free host memory
	delete[] rootArray_h;
	delete dDiffs_h;
}


vertexId_t* buildCaseArray(vertexId_t* diffs_h, length_t numRoots,
	vertexId_t& size, vertexId_t& adj, vertexId_t& nonadj,
	vertexId_t& adjRev, vertexId_t& nonadjRev)
{
	for (int k = 0; k < numRoots; k++)
	{
		if (diffs_h[k] == 1) {
			adj++;
		} else if (diffs_h[k] == -1) {
			adjRev++;
		} else if (diffs_h[k] > 1) {
			nonadj++;
		} else if (diffs_h[k] < -1) {
			nonadjRev++;
		}
	}

	// positions in the array of where to place each case type
	vertexId_t posA = 0;  // adj cases
	vertexId_t posB = posA + adj;  // adjRev cases
	vertexId_t posC = posB + adjRev;  // nonadj cases
	vertexId_t posD = posC + nonadj;  // nonadjRev cases

	size = posD + nonadjRev;

	vertexId_t* caseArray = new vertexId_t[size];

	for (int k = 0; k < numRoots; k++)
	{
		// printf("k: %d\tdiff: %d\n", k, diffs_h[k]);
		if (diffs_h[k] == 1) {
			caseArray[posA++] = k;
		} else if (diffs_h[k] == -1) {
			caseArray[posB++] = k;
		} else if (diffs_h[k] > 1) {
			caseArray[posC++] = k;
		} else if (diffs_h[k] < -1) {
			caseArray[posD++] = k;
		}
	}

	return caseArray;
}

} // cuStingerAlgs namespace