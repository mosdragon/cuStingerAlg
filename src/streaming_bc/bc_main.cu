#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <inttypes.h>

#include <math.h>


#include "utils.hpp"
#include "update.hpp"
#include "cuStinger.hpp"

#include <getopt.h>

#include "algs.cuh"
#include "streaming_bc/bc.cuh"
#include "streaming_bc/bc_tree.cuh"

using namespace cuStingerAlgs;
using namespace std;

#define CUDA(call, ...) do {                        \
        hipError_t _e = (call);                    \
        if (_e == hipSuccess) break;               \
        fprintf(stdout,                             \
                "CUDA runtime error: %s (%d)\n",    \
                hipGetErrorString(_e), _e);        \
        return -1;                                  \
    } while (0)

typedef struct
{
	bool streaming;
	bool approx;
	
	// number of vertices used. If approx, set here via CLI.
	int numRoots;
	bool verbose;  // print debug info
	int edgesToAdd;  // edges to add
	char *infile;
} program_options;


program_options options;

void printUsageInfo(char **argv)
{
	cout << "Usage: " << argv[0];
	cout << " -i <graph input file> [optional arguments]";
	cout << endl << endl;

	cout << "Options: " << endl;

	cout << "-v                      \tVerbose. Prints debug output";
	cout << " to stdout" << endl;

	cout << "-k <# of src nodes>     \tApproximate BC using a given";
	cout << " number of random source nodes" << endl;

	cout << "-t <# of nodes to add>  \tStreaming BC" << endl;
	cout << endl;
}


void parse_arguments(int argc, char **argv)
{
	int c;
	static struct option long_options[] =
	{
		{"help", no_argument, 0, 'h'},
		{"infile", required_argument, 0, 'i'},
		{"source_nodes", required_argument, 0, 'k'},
		{"stream", required_argument, 0, 't'},  // arg is # of edges to insert
		{"verbose", no_argument, 0,'v'},
		{0,0,0,0} // Terminate with null
	};

	int option_index = 0;

	while((c = getopt_long(argc, argv, "c:de::fg:hi:k:mn::opst:v",
		long_options, &option_index)) != -1)
	{
		switch(c)
		{
			case 'i':
				options.infile = optarg;
			break;

			case 'k':
				options.numRoots = atoi(optarg);
				options.approx = true;
			break;

			case 't':
				options.edgesToAdd = atoi(optarg);
				options.streaming = true;
			break;

			case 'v':
				options.verbose = true;
			break;

			case 'h':
				printUsageInfo(argv);
				exit(0);
			break;

			default: //Fatal error
				cerr << "Internal error parsing arguments." << endl;
				printUsageInfo(argv);
				exit(-1);
		}
	}

	//Handle required command line options here
	if(options.infile == NULL)
	{
		cerr << "Command line error: Graph input file is required.";
		cerr << " Use the -i switch." << endl;
		printUsageInfo(argv);
		exit(-1);
	}
	if(options.approx && (options.numRoots == -1 || options.numRoots < 1))
	{
		cerr << "Command line error: Approximation requested but no";
		cerr << " number of source nodes given. Defaulting to 128.";
		cerr << endl;
		options.numRoots = 128;
	}
	if(options.streaming && (options.edgesToAdd == -1))
	{
		cerr << "Command line error: Streaming requested but no";
		cerr << " number of insertions given. Defaulting to 5.";
		cerr << endl;
		options.edgesToAdd = 5;
	}
}


void generateEdgeUpdates(length_t nv, length_t numEdges, vertexId_t* edgeSrc, vertexId_t* edgeDst)
{
	cout << "Edge Updates: " << endl;
	for(int32_t e=0; e<numEdges; e++) {
		edgeSrc[e] = rand()%nv;
		edgeDst[e] = rand()%nv;

		cout << "Edge: (" << edgeSrc[e] << ", " << edgeDst[e] << ")";
		cout << endl;
	}
}


void rmat_edge (int64_t * iout, int64_t * jout, int SCALE, double A, double B, double C, double D)
{
  int64_t i = 0, j = 0;
  int64_t bit = ((int64_t) 1) << (SCALE - 1);

  while (1) {
    const double r =  ((double) rand() / (RAND_MAX));//dxor128(env);
    if (r > A) {                /* outside quadrant 1 */
      if (r <= A + B)           /* in quadrant 2 */
        j |= bit;
      else if (r <= A + B + C)  /* in quadrant 3 */
        i |= bit;
      else {                    /* in quadrant 4 */
        j |= bit;
        i |= bit;
      }
    }
    if (1 == bit)
      break;

    /*
      Assuming R is in (0, 1), 0.95 + 0.1 * R is in (0.95, 1.05).
      So the new probabilities are *not* the old +/- 10% but
      instead the old +/- 5%.
    */
    A *= (9.5 + ((double) rand() / (RAND_MAX))) / 10;
    B *= (9.5 + ((double) rand() / (RAND_MAX))) / 10;
    C *= (9.5 + ((double) rand() / (RAND_MAX))) / 10;
    D *= (9.5 + ((double) rand() / (RAND_MAX))) / 10;
    /* Used 5 random numbers. */

    {
      const double norm = 1.0 / (A + B + C + D);
      A *= norm;
      B *= norm;
      C *= norm;
    }
    /* So long as +/- are monotonic, ensure a+b+c+d <= 1.0 */
    D = 1.0 - (A + B + C);

    bit >>= 1;
  }
  /* Iterates SCALE times. */
  *iout = i;
  *jout = j;
}


void generateEdgeUpdatesRMAT(length_t nv, length_t numEdges, vertexId_t* edgeSrc, vertexId_t* edgeDst,double A, double B, double C, double D)
{
	int64_t src,dst;
	int scale = (int)log2(double(nv));
	for(int32_t e=0; e<numEdges; e++){
		rmat_edge(&src,&dst,scale, A,B,C,D);
		edgeSrc[e] = src;
		edgeDst[e] = dst;
	}
}

int main(const int argc, char **argv)
{
	parse_arguments(argc, argv);

	int device = 0;
    hipSetDevice(device);
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, device);

	bool isDimacs = false;
	bool isSNAP = false;
	bool isRmat = false;
	length_t nv, ne, *off;
	vertexId_t *adj;

	string filename(options.infile);

	isDimacs = filename.find(".graph")==string::npos?false:true;
	isSNAP   = filename.find(".txt")==string::npos?false:true;
	isRmat 	 = filename.find("kron")==string::npos?false:true;

	if(isDimacs){
	    readGraphDIMACS(options.infile, &off, &adj, &nv, &ne, isRmat);
	} else if(isSNAP){
	    readGraphSNAP(options.infile, &off, &adj, &nv, &ne);
	} else {
		cout << "Unknown graph type" << endl;
		exit(0);
	}

	// if not in approx mode, set numRoots to number of vertices
	if (!options.approx) {
		options.numRoots = nv;
	}

	cuStinger custing(defaultInitAllocater,defaultUpdateAllocater);

	cuStingerInitConfig hipInit;
	hipInit.initState = eInitStateCSR;
	hipInit.maxNV = nv + 1;
	hipInit.useVWeight = false;
	hipInit.isSemantic = false;  // Use edge types and vertex types
	hipInit.useEWeight = false;
	
	// CSR data
	hipInit.csrNV  = nv;
	hipInit.csrNE = ne;
	hipInit.csrOff  = off;
	hipInit.csrAdj  = adj;
	hipInit.csrVW  = NULL;
	hipInit.csrEW = NULL;

	custing.initializeCuStinger(hipInit);

	// Store betweenness centrality values here
	float *bc = new float[nv];
	for (int k = 0; k < nv; k++)
	{
		bc[k] = 0;
	}

	vertexId_t root = 0;
	int rootsVisited = 0;

	StreamingBC sbc(options.numRoots);
	sbc.Init(custing);
	sbc.setInputParameters(bc);

	hipEvent_t ce_start,ce_stop;
	start_clock(ce_start, ce_stop);

	sbc.Run(custing);

	float totalTime = end_clock(ce_start, ce_stop);
	cout << "Total time for Betweenness Centrality Computation: " << totalTime << endl;

	// Now, insert a random edge
	vertexId_t src = rand() % nv;
	vertexId_t dst = rand() % nv;
	
	cout << "About to insert edge: (" << src << ", " << dst << ")" << endl;
	start_clock(ce_start, ce_stop);
	sbc.InsertEdge(custing, src, dst);

	totalTime = end_clock(ce_start, ce_stop);
	cout << "Done inserting. Total time taken:  " << totalTime  << endl;

	if (options.verbose) {
		cout << "RESULTS: " << endl;

		for (int k = 0; k < nv; k++) {
			cout << "[ " << k  << " ]: " << bc[k] << endl;
		}
	}

	cout << "=======================================" << endl;
	cout << "Now doing brute force edge insertion" << endl;


	// Add that same edge into the graph and run static bc on it
	// TODO: figure out how to add edge
	length_t allocs;
	// auto bud = new BatchUpdateData(1, true, custing.nv);
	BatchUpdateData bud(1 , true, custing.nv);
	vertexId_t *srcs = bud.getSrc();
	vertexId_t *dsts = bud.getDst();
	srcs[0] = src;
	dsts[0] = dst;

	BatchUpdate bu = BatchUpdate(bud);

	custing.edgeInsertions(bu, allocs);

	float *bc_static = new float[nv];
	for (int k = 0; k < nv; k++) {
		bc_static[k] = 0;
	}	

	StreamingBC sbc2(options.numRoots);
	sbc2.Init(custing);
	sbc2.setInputParameters(bc_static);

	start_clock(ce_start, ce_stop);

	sbc2.Run(custing);

	totalTime = end_clock(ce_start, ce_stop);
	cout << "Done with static. Total time taken:  " << totalTime  << endl;

	bool same = true;
	for (int k = 0; k < nv; k++) {
		if (bc_static[k] != bc[k]) {
			same = false;
			break;
		}
	}

	cout << "Are they same?   :: " << (same?"true":"false") << endl;

	// free resources
	sbc.Reset();
	sbc.Release();

	// free resources
	sbc2.Reset();
	sbc2.Release();

	// Free memory
	custing.freecuStinger();

	free(off);
	free(adj);

	delete[] bc;
	delete[] bc_static;

    return 0;
}
